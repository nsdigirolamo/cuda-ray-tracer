#include "hip/hip_runtime.h"
#include "time.h"
#include <iostream>

#include "camera.hpp"
#include "image.hpp"

int main () {

    clock_t start = clock();

    Camera camera {
        {{ 0, 0, -20 }},
        1080,
        1920,
        90,
        {{ 0, 0, 0 }},
        5.0
    };

    Image image = camera.render(50, 50);

    image.writeToFile("render");

    clock_t end = clock();

    std::cout << "Completed in " << (end - start) / 1000000.0 << " seconds.\n";
}