#include "utils/rand_utils.hpp"

__device__ Point randomInUnitSphere (hiprandState* state) {

    double x, y, z;
    Point p;

    do {

        x = hiprand_uniform_double(state);
        y = hiprand_uniform_double(state);
        z = hiprand_uniform_double(state);

        p = {{ x, y, z }};

    } while (1 < p.length());

    return p;
}

__device__ Point randomOnUnitSphere (hiprandState* state) {

    return (UnitVector<3>)(randomInUnitSphere(state));
}
